
#include <hip/hip_runtime.h>
void setGrid(int n, dim3 &blockDim, dim3 &gridDim)
{
    // set your block dimensions and grid dimensions here
    // remember to edit these two parameters each time you change the block size
    gridDim.x = n / (blockDim.x * 2);
    gridDim.y = n / (blockDim.y * 2);
    if(n % (blockDim.x*2) != 0)
        gridDim.x++;
    if(n % (blockDim.y*2) != 0)
        gridDim.y++;
}
