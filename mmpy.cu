/*
 * Simplest matrix multiplication in CUDA
 *
 * Scott B. Baden, University of California, San Diego
 * April 2010
 *
 * We compute C = A * B
 *
 * This code assumes that the  matrices are square though there
 * are hooks to facilitate  extending the code to non-square matrices
 *
 */

// system includes
#include <stdio.h>
#include <assert.h>

#include <iostream>

//  include the kernel
#include "mmpy_kernel.cu"

#include "types.h"
#include "utils.h"

// External function definitions
void genMatrix( _DOUBLE_ *a, unsigned int m, unsigned int n);
void genMatrix_bt( _DOUBLE_ *a, _DOUBLE_ *b, unsigned int n);
void genMatrix_rand( _DOUBLE_ *a, _DOUBLE_ *b, unsigned int n);
void verify( _DOUBLE_ *c, unsigned int m, unsigned int n, _DOUBLE_ eps, const char *mesg);
void verify_bt( _DOUBLE_ *c, unsigned int n, const char *mesg);
void verify( _DOUBLE_ *c_d, _DOUBLE_ *c_h,  unsigned int m, unsigned int n, _DOUBLE_ eps, const char *mesg);
void verify_bt( _DOUBLE_ *c_d, _DOUBLE_ *c_h,  unsigned int n, const char *mesg);
void verify_bt( _DOUBLE_ *c_d, _DOUBLE_ *c_h,  unsigned int m, unsigned int n,  const char *mesg);
void verify_rand( _DOUBLE_ *a, _DOUBLE_ *b, _DOUBLE_ *c, unsigned int n);

void printMatrix( _DOUBLE_ *a, unsigned int m, unsigned int n);
void cmdLine(int argc, char *argv[], int& n, int& reps, int& ntx, int& nty, _DOUBLE_ & eps, int& do_host, int& prefer_l1, int& use_rand, int& use_bt, int& use_shm_double);
void perfString(int n, int ntx, int nty, int reps, double t_h, double gflops_h, double t_d, double gflops_d, int do_host, int prefer_l1, int use_rand, int use_bt, int use_shm_double);
// extern "C"{
    double getTime();
    double gflops(int n, int niter, double time);
//}
void matMulHost(_DOUBLE_ *, const _DOUBLE_ *, const _DOUBLE_ *, unsigned int, unsigned int);
void setGrid(int n, dim3 &blockDim, dim3 &gridDim);

int
main(int argc, char** argv) {
    // To improve repeatabilty of measurements taken on the device,
    // we multiply the number of reps by this scale factor
    // Adjust as needed
    const int SCALE = 10;

// Read in the command line elements
    int n, reps, ntx, nty, do_host, prefer_l1, use_rand, use_bt, use_shm_double;
    _DOUBLE_ eps;

    cmdLine(argc, argv, n, reps, ntx, nty, eps, do_host, prefer_l1, use_rand, use_bt, use_shm_double);

   // The thread geometry must evenly divide N
   /*if ((n % ntx != 0) || (n % nty != 0) )
   {
        printf("Thread geometry: %d x %d\n",ntx, nty);
        printf("The length of the thread geometry axis ");
        printf("[ %d x %d]\n",ntx, nty);
        printf("  nust divide N [%d] evenly\n",n);
        exit(-1);
   }
   */

    // Total amount of storage for entries
    unsigned int n2 = n*n*sizeof(_DOUBLE_);

    // Report on Device Characteristics
    int capability = ReportDevice();
#ifdef _DOUBLE
    int major = capability/100;
    int minor = capability%100;
    if ((major == 1) && (minor < 3)){
        printf("   *** You are running on a capability %d.%d device\n",major, minor);
	printf("       which does not support double precision arithmetic.\n");
	printf("       Recompile with single precision.\n\n");
	exit(-1);
    }
#endif

    // setup execution configurations
    int _ntx, _nty;
#if (!defined(BLOCKDIM_X) && !defined(BLOCKDIM_Y))
    _ntx = ntx;
    _nty = nty;
#else
    _ntx = BLOCKDIM_X;
    _nty = BLOCKDIM_Y;
#endif

    dim3 threads(_ntx, _nty,1);
    int numblocksX = n/_ntx;
    int numblocksY = n/_nty;

    if( n % _ntx != 0  )
        numblocksX++;

    if( n % _nty != 0  )
        numblocksY++;
 
    dim3 grid(numblocksX, numblocksY, 1);

    setGrid(n, threads, grid);

    // print configurations
    printf("n: %d, tx: %d, ty: %d, gridX: %d, gridY: %d, reps: %d, epsilon: %g\n\n", n, threads.x, threads.y, grid.x, grid.y, reps, eps);

  
#ifndef _DOUBLE
    printf("Using Single precision arithmetic\n\n");
#else
    printf("Using Double precision arithmetic\n\n");
#endif

    if (use_bt)
        printf("Using bidiagonal inputs\n");

    if (use_rand)
        printf("Using random inputs\n");

    if (do_host)
        printf("Doing host computation for comparison\n\n");

     printf("\n");

    // allocate an initialize host memory for A and B matrices
    _DOUBLE_ *h_A = (_DOUBLE_ *) malloc(n2);
    assert(h_A);
    _DOUBLE_ *h_B = (_DOUBLE_ *) malloc(n2);
    assert(h_B);
    if (use_bt){
        genMatrix_bt(h_A, h_B, n);
    }
    else if (use_rand){
        genMatrix_rand(h_A, h_B, n);
    }
    else{
        genMatrix(h_A, n, n);
        genMatrix(h_B, n, n);
    }

    if (n <= 8){
        cout << "\nA:\n";
        printMatrix( h_A, n,n);
        cout << "\nB:\n";
        printMatrix( h_B, n,n);
    }

    _DOUBLE_  *hostC;
    double t_host=0.0, gflops_h=0.0;
    if (do_host){
        // compute matrix product on the host
        hostC = (_DOUBLE_ *) malloc(n2);
        t_host = -getTime();
        for (int r=0; r< reps; r++)
            matMulHost(hostC, h_A, h_B, n, n);
        t_host += getTime();
        gflops_h = gflops(n, reps, t_host );
        printf("Host computation time: %f sec. [%f gflops]\n",t_host,gflops_h);

        // Verify host result
        if (use_bt)
            verify_bt( hostC,n, "Host result");
        else if (use_rand)
            cout << "Verfication of host result not supported for random matrices\n";
        else
            verify( hostC,n,n, eps, "Host result");

        if (n <= 8){
            printf("\nC:\n");
            printMatrix( hostC, n,n);
        }
    }

    // allocate device memory
    _DOUBLE_ *d_A, *d_B, *d_C;
    hipMalloc((void**) &d_A, n2);
    checkCUDAError("Error allocating device memory for matrix A");
    hipMalloc((void**) &d_B, n2);
    checkCUDAError("Error allocating device memory for matrix B");
    hipMalloc((void**) &d_C, n2);
    checkCUDAError("Error allocating device memory for matrix C");
    hipMemset((void **) d_A,-99,n2);
    checkCUDAError("Error initializing device memory matrix A");
    hipMemset((void **) d_B,-99,n2);
    checkCUDAError("Error initializing device memory matrix B");
    hipMemset((void **) d_C,0,n2);
    checkCUDAError("Error clearing device memory matrix C");

    // copy host memory to device
    hipMemcpy(d_A, h_A, n2, hipMemcpyHostToDevice);
    checkCUDAError("Error copying matrix A to device");
    hipMemcpy(d_B, h_B, n2, hipMemcpyHostToDevice);
    checkCUDAError("Error copying matrix B to device");


    // allocate host memory for the result
    _DOUBLE_  *h_C = (_DOUBLE_ *) malloc(n2);
    assert(h_C);


// If we set the preference for L1 cache, rather than
// shared memory, we may run slightly faster on devices that have the capability
    hipFuncCache_t Preference;
    if (prefer_l1){
        Preference = hipFuncCachePreferL1;
    }
    else{
        Preference = hipFuncCachePreferShared;
    } 
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(matMul),Preference);

    hipSharedMemConfig  shmPreference;
    if (use_shm_double){
      shmPreference = hipSharedMemBankSizeEightByte;
    }else{
      shmPreference = hipSharedMemBankSizeFourByte;
    }
    hipFuncSetSharedMemConfig(reinterpret_cast<const void*>( matMul), shmPreference);

    // Start the timer
#ifdef CUDA_TIMER
    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event) ;
    hipEventCreate(&stop_event);
#endif

#ifdef CUDA_TIMER
    hipEventRecord(start_event, 0);
    float t_device;
#else
    hipDeviceSynchronize();
    double t_device = -getTime();
#endif

    // execute the kernel
    for (int r=0; r< SCALE*reps; r++)
        matMul<<< grid, threads >>>(n, d_C, d_A, d_B);

#ifdef CUDA_TIMER
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&t_device, start_event, stop_event);
    t_device /= 1000.0;

#else
    // block until the device has finished
    hipDeviceSynchronize();
    // Stop the timer
    t_device +=getTime();
#endif

    checkCUDAError("Error in matrixMul kernel");

    // copy result from device to host
    hipMemcpy(h_C, d_C, n2, hipMemcpyDeviceToHost);
    checkCUDAError("Unable to retrieve result from device");



    double gflops_d = gflops(n, SCALE*reps, t_device );
    printf("Device computation time: %f sec. [%f gflops]\n",t_device,gflops_d);
    perfString(n, ntx, nty, reps, t_host, gflops_h, t_device, gflops_d, do_host, prefer_l1, use_rand, use_bt, use_shm_double);

    if (n <= 8){
        printf("\nC (device):\n");
        printMatrix( h_C, n,n);
    }
    // Verify the device result
    if (use_bt)
        verify_bt( h_C,n,"Device result");
    else if (use_rand)
        verify_rand( h_A, h_B, h_C, n);
    else
        verify( h_C,n,n, eps,"Device result");

    // But not for random matrices
    if (do_host)
        // Compare host and device results
        if (use_bt)
            verify_bt( h_C, hostC, n,"Device vs. host");
        else if (!use_rand)
            verify( h_C, hostC, n, n, eps,"Device vs. host");

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    if (do_host)
        free(hostC);

    assert(hipSuccess ==hipFree(d_A));
    assert(hipSuccess ==hipFree(d_B));
    assert(hipSuccess ==hipFree(d_C));

    hipDeviceReset();
}
